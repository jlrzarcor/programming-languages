#include "hip/hip_runtime.h"
//nvcc --compiler-options -Wall daxpy_cuda_c_3.cu funciones.c -o daxpy_cuda_c_3.out
// ./daxpy_cuda_c_3.out <dimension de vectores> <número de threads>

#include<stdio.h>

__global__ void daxpy(double *a, double *b, double *alpha, int *N){
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	int stride = blockDim.x*gridDim.x;
	int i;
		for(i=tid;i<*N;i+=stride)
			b[i] = b[i] + (*alpha)*a[i];

}


int main(int argc, char *argv[]){
	double *a, *b;
	double *device_a, *device_b;
	int i;
	double al=3.5;
	double *d_al;
	int N;
	int num_threads;
	int numSMs;
	int *d_N;
	double time_spent;

	//dimensiones
	N=atoi(argv[1]);

	num_threads=atoi(argv[2]);
	//alojando en host:
	a = (double *)calloc(N,sizeof(double));
	b = (double *)calloc(N,sizeof(double));

	//alojando en device
	hipMalloc((void **)&device_a, sizeof(double)*N);
	hipMalloc((void **)&device_b, sizeof(double)*N);
	hipMalloc((void **)&d_al, sizeof(double));
	hipMalloc((void **)&d_N, sizeof(int));
	//llenando los arreglos:
	for(i=0;i<N;i++){
		a[i]=i;
		b[i]=i*i;
	}
	//copiamos arreglos a, b a la GPU
	hipMemcpy(device_a,a,N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_b,b,N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_al,&al,sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_N,&N,sizeof(int), hipMemcpyHostToDevice);

	//número de SM's
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	//mandamos a llamar a daxpy:
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	clock_t begin = clock();
	daxpy<<<32*numSMs,num_threads>>>(device_a,device_b,d_al,d_N); //N bloques de 1 thread
    hipDeviceSynchronize();
    clock_t end = clock();
    //tiempo de cálculo:
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Tiempo de cálculo en la gpu %.5f\n", time_spent);
	//copia del resultado al arreglo b:
	hipMemcpy(b,device_b,N*sizeof(double),hipMemcpyDeviceToHost);
	
	hipFree(device_a);
	hipFree(device_b);
	hipFree(d_al);
	hipFree(d_N);
	return 0;
}




